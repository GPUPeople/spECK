#include "cusparse/include/cuSparseMultiply.h"
#include <hip/hip_runtime.h>


namespace cuSPARSE {
		template<>
		hipsparseStatus_t CUSPARSEAPI CuSparseTest<double>::cusparseMultiply(hipsparseHandle_t handle, hipsparseOperation_t transA, hipsparseOperation_t transB,
			int m, int n, int k, const hipsparseMatDescr_t descrA, int nnzA, const double *csrSortedValA, const int *csrSortedRowPtrA, const int *csrSortedColIndA,
			const hipsparseMatDescr_t descrB, int nnzB, const double *csrSortedValB, const int *csrSortedRowPtrB, const int *csrSortedColIndB,
			const hipsparseMatDescr_t descrC, double *csrSortedValC, const int *csrSortedRowPtrC, int *csrSortedColIndC){
			return hipsparseDcsrgemm(handle, transA, transB, m, n, k,
				descrA, nnzA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA,
				descrB, nnzB, csrSortedValB, csrSortedRowPtrB, csrSortedColIndB,
				descrC, csrSortedValC, csrSortedRowPtrC, csrSortedColIndC);
		}

		template<>
		hipsparseStatus_t CUSPARSEAPI CuSparseTest<float>::cusparseMultiply(hipsparseHandle_t handle, hipsparseOperation_t transA,	hipsparseOperation_t transB,	
			int m, int n, int k, const hipsparseMatDescr_t descrA, int nnzA, const float *csrSortedValA, const int *csrSortedRowPtrA, const int *csrSortedColIndA,
			const hipsparseMatDescr_t descrB, int nnzB, const float *csrSortedValB, const int *csrSortedRowPtrB,	const int *csrSortedColIndB,
			const hipsparseMatDescr_t descrC, float *csrSortedValC, const int *csrSortedRowPtrC,	int *csrSortedColIndC){
			return hipsparseScsrgemm(handle, transA, transB, m, n, k,
				descrA, nnzA, csrSortedValA, csrSortedRowPtrA, csrSortedColIndA,
				descrB, nnzB, csrSortedValB, csrSortedRowPtrB, csrSortedColIndB,
				descrC, csrSortedValC, csrSortedRowPtrC, csrSortedColIndC);
		}

		template<>
		hipsparseStatus_t CUSPARSEAPI CuSparseTest<float>::cusparseTranspose(hipsparseHandle_t handle, int m, int n, int nnz,
			const float  *csrSortedVal,	const int *csrSortedRowPtr, const int *csrSortedColInd,
			float *cscSortedVal, int *cscSortedRowInd, int *cscSortedColPtr, hipsparseAction_t copyValues, hipsparseIndexBase_t idxBase)
		{
			return hipsparseScsr2csc(handle, m, n, nnz, csrSortedVal, csrSortedRowPtr, csrSortedColInd, cscSortedVal,
				cscSortedRowInd, cscSortedColPtr, copyValues, idxBase);
		}

		template<>
		hipsparseStatus_t CUSPARSEAPI CuSparseTest<double>::cusparseTranspose(hipsparseHandle_t handle,  int m, int n, int nnz,
			const double  *csrSortedVal, const int *csrSortedRowPtr, const int *csrSortedColInd,
			double *cscSortedVal, int *cscSortedRowInd, int *cscSortedColPtr, hipsparseAction_t copyValues, hipsparseIndexBase_t idxBase)
		{
			return hipsparseDcsr2csc(handle, m, n, nnz, csrSortedVal, csrSortedRowPtr, csrSortedColInd, cscSortedVal,
				cscSortedRowInd, cscSortedColPtr, copyValues, idxBase);
		}

	template <typename DataType>
	float CuSparseTest<DataType>::Multiply(const dCSR<DataType>& A, const dCSR<DataType>& B, dCSR<DataType>& matOut, uint32_t& cusparse_nnz)
	{
		int nnzC;
		int *nnzTotalDevHostPtr = &nnzC;
		float duration;
		int m, n, k;
		m = A.rows;
		n = B.cols;
		k = A.cols;
		// matOut.reset();

		hipEvent_t start, stop;
		hipEventCreate(&start);
		hipEventCreate(&stop);

		// ############################
		hipEventRecord(start);
		// ############################

		// Allocate memory for row indices
		if(matOut.rows != A.rows)
		{
			if (matOut.row_offsets != nullptr)
				hipFree(matOut.row_offsets);

			hipMalloc(&(matOut.row_offsets), sizeof(uint32_t) * (A.rows + 1));
		}

		// Precompute number of nnz in C
		checkCuSparseError(hipsparseXcsrgemmNnz(
			handle,
			HIPSPARSE_OPERATION_NON_TRANSPOSE,
			HIPSPARSE_OPERATION_NON_TRANSPOSE,
			m, n, k,
			descr, A.nnz, reinterpret_cast<const int*>(A.row_offsets), reinterpret_cast<const int*>(A.col_ids),
			descrB, B.nnz, reinterpret_cast<const int*>(B.row_offsets), reinterpret_cast<const int*>(B.col_ids),
			descrC, reinterpret_cast<int*>(matOut.row_offsets), nnzTotalDevHostPtr), "cuSparse: Precompute failed"
		);

		cusparse_nnz = nnzC;

		// Allocate rest of memory
		if(nnzC != matOut.nnz)
		{
			if (matOut.col_ids != nullptr)
				hipFree(matOut.col_ids);
			if (matOut.data != nullptr)
				hipFree(matOut.data);

			hipMalloc(&(matOut.col_ids), sizeof(uint32_t) * nnzC);
			hipMalloc(&(matOut.data), sizeof(DataType) * nnzC);
		}
		
		// Compute SpGEMM
		checkCuSparseError(cusparseMultiply(
			handle,
			HIPSPARSE_OPERATION_NON_TRANSPOSE,
			HIPSPARSE_OPERATION_NON_TRANSPOSE,
			m, n, k,
			descr, A.nnz, reinterpret_cast<const DataType*>(A.data), reinterpret_cast<const int*>(A.row_offsets), reinterpret_cast<const int*>(A.col_ids),
			descrB, B.nnz, reinterpret_cast<const DataType*>(B.data), reinterpret_cast<const int*>(B.row_offsets), reinterpret_cast<const int*>(B.col_ids),
			descrC, reinterpret_cast<DataType*>(matOut.data), reinterpret_cast<int*>(matOut.row_offsets), reinterpret_cast<int*>(matOut.col_ids)),
			"cuSparse: SpGEMM failed");

		matOut.nnz = nnzC;
		matOut.rows = m;
		matOut.cols = n;

		// ############################
		hipDeviceSynchronize();
		hipEventRecord(stop);
		hipEventSynchronize(stop);
		// ############################

		hipEventElapsedTime(&duration, start, stop);

		return duration;
	}

	template float CuSparseTest<float>::Multiply(const dCSR<float>& A, const dCSR<float>& B, dCSR<float>& matOut, uint32_t& cusparse_nnz);
	template float CuSparseTest<double>::Multiply(const dCSR<double>& A, const dCSR<double>& B, dCSR<double>& matOut, uint32_t& cusparse_nnz);

	template <typename DataType>
	void CuSparseTest<DataType>::Transpose(const dCSR<DataType>& A, dCSR<DataType>& AT)
	{
		AT.alloc(A.cols, A.rows, A.nnz);

		checkCuSparseError(cusparseTranspose(handle, A.rows, A.cols, A.nnz,
			reinterpret_cast<const DataType*>(A.data), reinterpret_cast<const int*>(A.row_offsets), reinterpret_cast<const int*>(A.col_ids),
			reinterpret_cast<DataType*>(AT.data), reinterpret_cast<int*>(AT.col_ids), reinterpret_cast<int*>(AT.row_offsets),
			HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO), "transpose failed");
	}

	template	void CuSparseTest<float>::Transpose(const dCSR<float>& A, dCSR<float>& AT);
	template	void CuSparseTest<double>::Transpose(const dCSR<double>& A, dCSR<double>& AT);
}